#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "render_cuda.h"

#define WIDTH 1200
#define HEIGHT 900

# define SAMPLES_PER_PIXEL 256
# define SHININESS 64.0f

#define CUDA_CHECK(err) { \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
}

// GPU version of light structure without 'next' pointer
typedef struct s_light_gpu
{
	t_vector	position;
	float		ratio;
	t_color		color;
}	t_light_gpu;

// 简单的线性同余伪随机数生成器，适用于 GPU
__device__ unsigned int cuda_rand(unsigned int *seed)
{
    *seed = (*seed * 1103515245U + 12345U) & 0x7fffffffU;
    return *seed;
}

// 生成 [0, 1) 范围内的随机浮点数
__device__ float cuda_randf(unsigned int *seed)
{
    return (float)cuda_rand(seed) / (float)0x80000000U;
}

__device__ t_vector	vector_init(float x, float y, float z)
{
	t_vector	res;

	res.x = x;
	res.y = y;
	res.z = z;
	return (res);
}

__device__ t_vector	vector_add(t_vector v1, t_vector v2)
{
	t_vector	res;

	res.x = v1.x + v2.x;
	res.y = v1.y + v2.y;
	res.z = v1.z + v2.z;
	return (res);
}

__device__ t_vector	vector_sub(t_vector v1, t_vector v2)
{
	t_vector	res;

	res.x = v1.x - v2.x;
	res.y = v1.y - v2.y;
	res.z = v1.z - v2.z;
	return (res);
}

__device__ t_vector	vector_mult(t_vector v, float scalar)
{
	t_vector	res;

	res.x = v.x * scalar;
	res.y = v.y * scalar;
	res.z = v.z * scalar;
	return (res);
}

__device__ t_vector	vector_div(t_vector v, float scalar)
{
	t_vector	res;

	res.x = v.x / scalar;
	res.y = v.y / scalar;
	res.z = v.z / scalar;
	return (res);
}

__device__ float	vector_length(t_vector v)
{
	float	res;

	res = sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
	return (res);
}

__device__ float	vector_length_sq(t_vector v)
{
	float	res;

	res = v.x * v.x + v.y * v.y + v.z * v.z;
	return (res);
}

__device__ t_vector	vector_normalize(t_vector v)
{
	t_vector	res;
	float		v_length;

	v_length = vector_length(v);
	if (v_length == 0)
		return (vector_init(0, 0, 0));
	res = vector_div(v, v_length);
	return (res);
}

__device__ float	vector_dot(t_vector v1, t_vector v2)
{
	float	res;

	res = (v1.x * v2.x) + (v1.y * v2.y) + (v1.z * v2.z);
	return (res);
}

__device__ t_vector	vector_cross(t_vector v1, t_vector v2)
{
	t_vector	res;

	res.x = (v1.y * v2.z) - (v1.z * v2.y);
	res.y = (v1.z * v2.x) - (v1.x * v2.z);
	res.z = (v1.x * v2.y) - (v1.y * v2.x);
	return (res);
}

__device__ t_color	color_init(int r, int g, int b)
{
	t_color	res;

	res.r = (float)r / 255.0f;
	res.g = (float)g / 255.0f;
	res.b = (float)b / 255.0f;
	return (res);
}

__device__ t_color	color_init_d(float r, float g, float b)
{
	t_color	res;

	res.r = r;
	res.g = g;
	res.b = b;
	return (res);
}

__device__ t_color	color_add(t_color a, t_color b)
{
	t_color	res;

	res.r = a.r + b.r;
	res.g = a.g + b.g;
	res.b = a.b + b.b;
	return (res);
}

__device__ t_color	color_div(t_color a, float scalar)
{
	t_color	res;

	res.r = a.r / scalar;
	res.g = a.g / scalar;
	res.b = a.b / scalar;
	return (res);
}

__device__ int	color_to_int(t_color color)
{
	int	r;
	int	g;
	int	b;

	r = (int)(fminf(color.r, 1.0f) * 255.0f);
	g = (int)(fminf(color.g, 1.0f) * 255.0f);
	b = (int)(fminf(color.b, 1.0f) * 255.0f);
	if (r < 0)
		r = 0;
	if (g < 0)
		g = 0;
	if (b < 0)
		b = 0;
	return (r << 16 | g << 8 | b);
}


__device__ float	hit_sphere_gpu(const t_sphere *sp, const t_ray *ray)
{
	t_vector	oc;
	float		a;
	float		b;
	float		c;
	float		discriminant;

	oc = vector_sub(ray->origin, sp->center);
	a = vector_dot(ray->direction, ray->direction);
	b = 2.0f * vector_dot(oc, ray->direction);
	c = vector_dot(oc, oc) - sp->radius * sp->radius;
	discriminant = b * b - 4 * a * c;
	if (discriminant < 0)
		return (-1.0f);
	else
	{
		float t = (-b - sqrtf(discriminant)) / (2.0f * a);
		if (t > EPSILON)
			return (t);
		t = (-b + sqrtf(discriminant)) / (2.0f * a);
		if (t > EPSILON)
			return (t);
		return (-1.0f);
	}
}

__device__ t_vector	sphere_normal_at_gpu(t_sphere *sp, t_vector p)
{
	t_vector	normal;

	normal = vector_sub(p, sp->center);
	return (vector_normalize(normal));
}

__device__ float	hit_plane_gpu(t_plane *plane, t_ray *ray)
{
	float		t;
	float		dn;
	t_vector	op;

	op = vector_sub(plane->point, ray->origin);
	dn = vector_dot(ray->direction, plane->normal);
	if (fabsf(dn) < EPSILON)
		return (-1.0f);
	t = vector_dot(op, plane->normal) / dn;
	if (t > EPSILON)
		return (t);
	return (-1.0f);
}


static __device__ float	hit_main(t_cylinder *cy, t_ray *ray, t_cy_info *info)
{
	float	t1;
	float	t2;
	float	m1;
	float	m2;

	if (info->discri < 0)
		return (-1.0f);
	t1 = (-info->b - sqrtf(info->discri)) / (2 * info->a);
	t2 = (-info->b + sqrtf(info->discri)) / (2 * info->a);
	m1 = vector_dot(ray->direction, info->normal) * t1
		+ vector_dot(info->oc, info->normal);
	m2 = vector_dot(ray->direction, info->normal) * t2
		+ vector_dot(info->oc, info->normal);
	if (t1 > EPSILON && m1 >= 0 && m1 <= cy->height)
		return (t1);
	if (t2 > EPSILON && m2 >= 0 && m2 <= cy->height)
		return (t2);
	return (-1.0f);
}

static __device__ float	hit_caps2(t_plane *cap, t_ray *ray, t_cy_info *info)
{
	float		t;
	t_vector	p;
	float		radius_sq;

	radius_sq = info->radius * info->radius;
	t = hit_plane_gpu(cap, ray);
	if (t > 0)
	{
		p = vector_add(ray->origin, vector_mult(ray->direction, t));
		if (vector_length_sq(vector_sub(p, cap->point)) <= radius_sq)
			return (t);
	}
	return (-1.0f);
}

static __device__ float	hit_caps(t_cylinder *cy, t_ray *ray, t_cy_info *info)
{
	t_plane		cap;
	float		t1;
	float		t2;

	cap.point = vector_add(cy->center, vector_mult(info->normal, cy->height));
	cap.normal = info->normal;
	t1 = hit_caps2(&cap, ray, info);
	cap.point = cy->center;
	cap.normal = vector_mult(info->normal, -1);
	t2 = hit_caps2(&cap, ray, info);
	if (t1 > 0 && (t2 < 0 || t1 < t2))
		return (t1);
	return (t2);
}

__device__ float	hit_cylinder_gpu(t_cylinder *cy, t_ray *ray)
{
	t_cy_info	info;
	float		side_t;
	float		caps_t;

	info.normal = vector_normalize(cy->normal);
	info.radius = cy->diameter / 2.0f;
	info.oc = vector_sub(ray->origin, cy->center);
	info.a = vector_dot(ray->direction, ray->direction)
		- powf(vector_dot(ray->direction, info.normal), 2);
	info.b = 2 * (vector_dot(ray->direction, info.oc)
			- (vector_dot(ray->direction, info.normal)
				* vector_dot(info.oc, info.normal)));
	info.c = vector_dot(info.oc, info.oc)
		- powf(vector_dot(info.oc, info.normal), 2)
		- info.radius * info.radius;
	info.discri = info.b * info.b - 4 * info.a * info.c;
	side_t = hit_main(cy, ray, &info);
	caps_t = hit_caps(cy, ray, &info);
	if (side_t > 0 && (caps_t < 0 || side_t < caps_t))
		return (side_t);
	return (caps_t);
}

__device__ t_vector	cylinder_normal_at_gpu(t_cylinder *cy, t_vector point)
{
	t_vector	normal;
	t_vector	oc;
	float		m;

	normal = vector_normalize(cy->normal);
	oc = vector_sub(point, cy->center);
	m = vector_dot(oc, normal);
	if (m > cy->height - EPSILON)
		return (normal);
	if (m < EPSILON)
		return (vector_mult(normal, -1));
	return (vector_normalize(vector_sub(oc, vector_mult(normal, m))));
}

static __device__ void	fill_co_info_gpu(t_cone *co, t_ray *ray, t_co_info *info)
{
	float	cos2;

	cos2 = cosf(co->angle) * cosf(co->angle);
	info->a = info->dv * info->dv - cos2;
	info->b = 2 * (info->dv * info->ocv
			- cos2 * vector_dot(ray->direction, info->oc));
	info->c = info->ocv * info->ocv
		- cos2 * vector_dot(info->oc, info->oc);
	info->discr = info->b * info->b - 4 * info->a * info->c;
}

static __device__ float	hit_cone_body_gpu(t_cone *co, t_ray *ray)
{
	float		t1;
	float		t2;
	float		m1;
	float		m2;
	t_co_info	info;

	info.oc = vector_sub(ray->origin, co->center);
	info.dv = vector_dot(ray->direction, co->normal);
	info.ocv = vector_dot(info.oc, co->normal);
	fill_co_info_gpu(co, ray, &info);
	if (info.discr < 0)
		return (-1.0f);
	t1 = (-info.b - sqrtf(info.discr)) / (2 * info.a);
	m1 = info.ocv + t1 * info.dv;
	if (t1 <= EPSILON || m1 < 0 || m1 > co->height)
		t1 = -1.0f;
	t2 = (-info.b + sqrtf(info.discr)) / (2 * info.a);
	m2 = info.ocv + t2 * info.dv;
	if (t2 <= EPSILON || m2 < 0 || m2 > co->height)
		t2 = -1.0f;
	if (t1 > 0 && t2 > 0)
		return (fminf(t1, t2));
	if (t1 > 0)
		return (t1);
	return (t2);
}

static __device__ float	hit_cone_cap_gpu(t_cone *co, t_ray *ray)
{
	t_plane		cap;
	t_vector	base;
	t_vector	p;
	float		t;
	float		r;

	base = vector_add(co->center,
			vector_mult(vector_normalize(co->normal), co->height));
	cap.point = base;
	cap.normal = co->normal;
	t = hit_plane_gpu(&cap, ray);
	if (t > 0)
	{
		p = vector_add(ray->origin, vector_mult(ray->direction, t));
		r = tanf(co->angle) * co->height;
		if (vector_length_sq(vector_sub(p, base)) <= r * r)
			return (t);
	}
	return (-1.0f);
}

__device__ float	hit_cone_gpu(t_cone *co, t_ray *ray)
{
	float	t_body;
	float	t_cap;

	t_body = hit_cone_body_gpu(co, ray);
	t_cap = hit_cone_cap_gpu(co, ray);
	if (t_body > 0 && (t_cap < 0 || t_body < t_cap))
		return (t_body);
	return (t_cap);
}

static __device__ t_vector	cone_normal_2_gpu(t_cone *co, t_vector oc, t_vector v, float m)
{
	t_vector	radial;
	t_vector	unit_radial;
	float		cos_angle;
	float		sin_angle;
	t_vector	normal;

	radial = vector_sub(oc, vector_mult(v, m));
	if (vector_length_sq(radial) < EPSILON * EPSILON)
		return (vector_init(1, 0, 0));
	unit_radial = vector_normalize(radial);
	cos_angle = cosf(co->angle);
	sin_angle = sinf(co->angle);
	normal = vector_add(
		vector_mult(unit_radial, sin_angle),
		vector_mult(v, -cos_angle)
	);
	return (vector_normalize(normal));
}

__device__ t_vector	cone_normal_at_gpu(t_cone *co, t_vector point)
{
	t_vector	v;
	t_vector	oc;
	float		m;
	float		k;
	t_vector	normal;

	v = vector_normalize(co->normal);
	oc = vector_sub(point, co->center);
	m = vector_dot(oc, v);
	if (fabsf(m - co->height) < EPSILON)
		return (v);
	k = tanf(co->angle);
	if (m < EPSILON)
		return (cone_normal_2_gpu(co, oc, v, m));
	normal = vector_sub(oc, vector_mult(v, m * (1 + k * k)));
	return (vector_normalize(normal));
}

static __device__ void	cal_point_and_normal_gpu(t_hit_record *record, t_ray *ray, t_minirt_gpu *rt)
{
	record->point = vector_add(ray->origin,
			vector_mult(ray->direction, record->t));

	if (record->type == SPHERE)
	{
		t_sphere *sp = &rt->spheres[record->obj_idx];
		record->normal = sphere_normal_at_gpu(sp, record->point);
	}
	else if (record->type == PLANE)
	{
		t_plane *pl = &rt->planes[record->obj_idx];
		record->normal = pl->normal;
	}
	else if (record->type == CYLINDER)
	{
		t_cylinder *cy = &rt->cylinders[record->obj_idx];
		record->normal = cylinder_normal_at_gpu(cy, record->point);
	}
	else if (record->type == CONE)
	{
		t_cone *co = &rt->cones[record->obj_idx];
		record->normal = cone_normal_at_gpu(co, record->point);
	}
}

__device__ t_hit_record	trace_ray_gpu(t_minirt_gpu *rt, t_ray *ray)
{
	t_hit_record	record;
	float			t;

	record.hit = false;
	record.t = 1e30f;

	for (int i = 0; i < rt->num_sp; ++i)
	{
		t = hit_sphere_gpu(&rt->spheres[i], ray);
		if (t > EPSILON && t < record.t)
		{
			record.hit = true;
			record.t = t;
			record.obj_color = rt->spheres[i].color;
			record.type = SPHERE;
			record.obj_idx = i;
		}
	}
	for (int i = 0; i < rt->num_pl; ++i)
	{
		t = hit_plane_gpu(&rt->planes[i], ray);
		if (t > EPSILON && t < record.t)
		{
			record.hit = true;
			record.t = t;
			record.obj_color = rt->planes[i].color;
			record.type = PLANE;
			record.obj_idx = i;
		}
	}
	for (int i = 0; i < rt->num_cy; ++i)
	{
		t = hit_cylinder_gpu(&rt->cylinders[i], ray);
		if (t > EPSILON && t < record.t)
		{
			record.hit = true;
			record.t = t;
			record.obj_color = rt->cylinders[i].color;
			record.type = CYLINDER;
			record.obj_idx = i;
		}
	}
	for (int i = 0; i < rt->num_co; ++i)
	{
		t = hit_cone_gpu(&rt->cones[i], ray);
		if (t > EPSILON && t < record.t)
		{
			record.hit = true;
			record.t = t;
			record.obj_color = rt->cones[i].color;
			record.type = CONE;
			record.obj_idx = i;
		}
	}
	if (record.hit)
		cal_point_and_normal_gpu(&record, ray, rt);
	return (record);
}

// render_cuda/cuda_renderer.cu

__device__ bool is_in_shadow_gpu(const t_minirt_gpu *scene, t_vector hit_point, t_vector light_pos)
{
	t_ray	shadow_ray;
	float	dist_to_light;
	float	t;

    // 创建射向光源的阴影光线
	shadow_ray.direction = vector_normalize(vector_sub(light_pos, hit_point));
    // 将起点沿光线方向稍微偏移，避免自相交
	shadow_ray.origin = vector_add(hit_point, vector_mult(shadow_ray.direction, EPSILON));
	
    dist_to_light = vector_length(vector_sub(light_pos, hit_point));

	// 检查与所有对象的相交情况
	// 遍历球体
	for (int i = 0; i < scene->num_sp; ++i)
	{
		t = hit_sphere_gpu(&scene->spheres[i], &shadow_ray);
		if (t > 0.0f && t < dist_to_light)
			return (true); // 被遮挡
	}
	// 遍历平面
	for (int i = 0; i < scene->num_pl; ++i)
	{
		t = hit_plane_gpu(&scene->planes[i], &shadow_ray);
		if (t > 0.0f && t < dist_to_light)
			return (true); // 被遮挡
	}
	// 遍历圆柱体
	for (int i = 0; i < scene->num_cy; ++i)
	{
		t = hit_cylinder_gpu(&scene->cylinders[i], &shadow_ray);
		if (t > 0.0f && t < dist_to_light)
			return (true); // 被遮挡
	}
	// 遍历圆锥体
	for (int i = 0; i < scene->num_co; ++i)
	{
		t = hit_cone_gpu(&scene->cones[i], &shadow_ray);
		if (t > 0.0f && t < dist_to_light)
			return (true); // 被遮挡
	}

	return (false); // 未被遮挡
}

// render_cuda/cuda_renderer.cu

// 移植 calculate_ambient_light
__device__ static t_color calculate_ambient_light_gpu(t_ambient ambient, t_color obj_color)
{
	t_color	result;
	result.r = ambient.ratio * ambient.color.r * obj_color.r;
	result.g = ambient.ratio * ambient.color.g * obj_color.g;
	result.b = ambient.ratio * ambient.color.b * obj_color.b;
	return (result);
}

// 移植 calculate_diffuse_light
__device__ static t_color calculate_diffuse_light_gpu(t_light_gpu light, t_vector hit_point,
								t_vector normal, t_color obj_color)
{
	t_vector	light_dir;
	float		light_intensity;
	t_color		result;

	light_dir = vector_normalize(vector_sub(light.position, hit_point));
	light_intensity = fmaxf(0.0f, vector_dot(normal, light_dir)); // fmaxf确保不为负
	light_intensity *= light.ratio;
	
	result.r = light_intensity * obj_color.r * light.color.r;
	result.g = light_intensity * obj_color.g * light.color.g;
	result.b = light_intensity * obj_color.b * light.color.b;
	return (result);
}

// 添加镜面反射计算
__device__ static t_color calculate_specular_light_gpu(t_camera camera, t_light_gpu light, 
								t_vector hit_point, t_vector normal)
{
	t_vector	light_dir;
	t_vector	view_dir;
	t_vector	reflect_dir;
	float		spec_intensity;
	t_color		result;

	light_dir = vector_normalize(vector_sub(light.position, hit_point));
	view_dir = vector_normalize(vector_sub(camera.viewpoint, hit_point));
	reflect_dir = vector_sub(
			vector_mult(normal, 2.0f * vector_dot(normal, light_dir)), light_dir);
	spec_intensity = powf(fmaxf(vector_dot(view_dir, reflect_dir), 0.0f), SHININESS);
	
	result.r = light.ratio * spec_intensity * light.color.r;
	result.g = light.ratio * spec_intensity * light.color.g;
	result.b = light.ratio * spec_intensity * light.color.b;
	return (result);
}


__device__ t_color combine_light_gpu(const t_minirt_gpu *scene, const t_hit_record *hit)
{
	t_color	ambient_color;
	t_color	diffuse_color;
	t_color	specular_color;
	t_color	final_color;
	t_light_gpu *lights_array = (t_light_gpu *)scene->lights;

	// 计算环境光
	ambient_color = calculate_ambient_light_gpu(scene->ambient, hit->obj_color);
	final_color = ambient_color;

	// 遍历所有光源
	for (int i = 0; i < scene->num_lights; ++i)
	{
		// 检查阴影并计算漫反射光和镜面反射光
		if (!is_in_shadow_gpu(scene, hit->point, lights_array[i].position))
		{
			diffuse_color = calculate_diffuse_light_gpu(lights_array[i], hit->point,
					hit->normal, hit->obj_color);
			specular_color = calculate_specular_light_gpu(scene->camera, lights_array[i], 
					hit->point, hit->normal);
			final_color = color_add(final_color, diffuse_color);
			final_color = color_add(final_color, specular_color);
		}
	}

	// 颜色裁剪，防止超过1.0
	final_color.r = fminf(1.0f, final_color.r);
	final_color.g = fminf(1.0f, final_color.g);
	final_color.b = fminf(1.0f, final_color.b);

	return (final_color);
}


//global
// render_cuda/cuda_renderer.cu (续)

__global__ void render_kernel(int *pixel_buffer, t_minirt_gpu *scene)
{
    // 计算当前线程对应的像素坐标 (x, y)
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // 确保坐标在图像范围内
    if (x >= WIDTH || y >= HEIGHT)
        return;

    // 为每个线程初始化随机种子
    unsigned int seed = ((y * WIDTH + x) * 1103515245U + 12345U) ^ 0xdeadbeef;

    // 实现多重采样抗锯齿 (MSAA)
    t_color pixel_color = {0.0f, 0.0f, 0.0f};

    for (int s = 0; s < SAMPLES_PER_PIXEL; ++s)
    {
        // 在像素内生成随机偏移，实现抗锯齿
        float rand_offset_x = cuda_randf(&seed);
        float rand_offset_y = cuda_randf(&seed);
        
        float screen_x = (2.0f * (x + rand_offset_x) / WIDTH - 1.0f) * scene->camera.half_width;
        float screen_y = -(2.0f * (y + rand_offset_y) / HEIGHT - 1.0f) * scene->camera.half_height;

        t_vector screen_point = scene->camera.viewpoint;
        screen_point = vector_add(screen_point, vector_mult(scene->camera.u, screen_x));
        screen_point = vector_add(screen_point, vector_mult(scene->camera.v, screen_y));
        screen_point = vector_add(screen_point, vector_mult(scene->camera.w, -1.0f));

        t_ray ray;
        ray.origin = scene->camera.viewpoint;
        ray.direction = vector_normalize(vector_sub(screen_point, scene->camera.viewpoint));

        // 追踪光线并计算颜色
        t_hit_record hit = trace_ray_gpu(scene, &ray);
		t_color final_color_for_sample = {0.0f, 0.0f, 0.0f};
        if (hit.hit) {
			cal_point_and_normal_gpu(&hit, &ray, scene);
			final_color_for_sample = combine_light_gpu(scene, &hit);
		}
		pixel_color = color_add(pixel_color, final_color_for_sample);
    }
    
    // 平均所有采样的颜色值
    pixel_color = color_div(pixel_color, SAMPLES_PER_PIXEL);
    
    // 将最终颜色写入输出缓冲区
    int color_int = color_to_int(pixel_color);
    pixel_buffer[y * WIDTH + x] = color_int;
}


// run in cpu
static void	prepare_gpu(t_minirt *rt, t_minirt_gpu *host, t_minirt_gpu *device)
{
	t_object	*curr;
	t_light		*light_curr;

	host->ambient = rt->ambient;
	host->camera = rt->camera;

	// 计算light数量
	host->num_lights = 0;
	light_curr = rt->light;
	while (light_curr)
	{
		host->num_lights++;
		light_curr = light_curr->next;
	}

	// 计算对象数量
	host->num_sp = 0;
	host->num_pl = 0;
	host->num_cy = 0;
	host->num_co = 0;

	curr = rt->object;
	while (curr)
	{
		if (curr->type == SPHERE) host->num_sp++;
		else if (curr->type == PLANE) host->num_pl++;
		else if (curr->type == CYLINDER) host->num_cy++;
		else if (curr->type == CONE) host->num_co++;
		curr = curr->next;
	}
	
	// 初始化指针为NULL
	device->spheres = NULL;
	device->planes = NULL;
	device->cylinders = NULL;
	device->cones = NULL;
	device->lights = NULL;
	
	// 分配GPU内存
	if (host->num_sp > 0)
        CUDA_CHECK(hipMalloc(&device->spheres, sizeof(t_sphere) * host->num_sp));
    if (host->num_pl > 0)
        CUDA_CHECK(hipMalloc(&device->planes, sizeof(t_plane) * host->num_pl));
    if (host->num_cy > 0)
        CUDA_CHECK(hipMalloc(&device->cylinders, sizeof(t_cylinder) * host->num_cy));
    if (host->num_co > 0)
        CUDA_CHECK(hipMalloc(&device->cones, sizeof(t_cone) * host->num_co));
    if (host->num_lights > 0)
        CUDA_CHECK(hipMalloc(&device->lights, sizeof(t_light_gpu) * host->num_lights));

	// 为临时主机数组分配内存
	t_sphere* spheres_h = NULL;
	t_plane* planes_h = NULL;
	t_cylinder* cylinders_h = NULL;
	t_cone* cones_h = NULL;
	t_light_gpu* lights_h = NULL;
	
	if (host->num_sp > 0)
		spheres_h = (t_sphere*)malloc(sizeof(t_sphere) * host->num_sp);
	if (host->num_pl > 0)
		planes_h = (t_plane*)malloc(sizeof(t_plane) * host->num_pl);
	if (host->num_cy > 0)
		cylinders_h = (t_cylinder*)malloc(sizeof(t_cylinder) * host->num_cy);
	if (host->num_co > 0)
		cones_h = (t_cone*)malloc(sizeof(t_cone) * host->num_co);
	if (host->num_lights > 0)
		lights_h = (t_light_gpu*)malloc(sizeof(t_light_gpu) * host->num_lights);

	// 复制对象数据
	int sp_idx = 0, pl_idx = 0, cy_idx = 0, co_idx = 0;
	curr = rt->object;
	while (curr)
	{
		if (curr->type == SPHERE && spheres_h) 
			spheres_h[sp_idx++] = *(t_sphere*)curr->obj;
		if (curr->type == PLANE && planes_h) 
			planes_h[pl_idx++] = *(t_plane*)curr->obj;
		if (curr->type == CYLINDER && cylinders_h) 
			cylinders_h[cy_idx++] = *(t_cylinder*)curr->obj;
		if (curr->type == CONE && cones_h) 
			cones_h[co_idx++] = *(t_cone*)curr->obj;
		curr = curr->next;
	}

	// 复制light数据 - 转换为GPU格式（去掉next指针）
	int light_idx = 0;
	light_curr = rt->light;
	while (light_curr && lights_h)
	{
		lights_h[light_idx].position = light_curr->position;
		lights_h[light_idx].ratio = light_curr->ratio;
		lights_h[light_idx].color = light_curr->color;
		light_idx++;
		light_curr = light_curr->next;
	}
	
	// 将数据复制到GPU
	if (host->num_sp > 0 && spheres_h)
		CUDA_CHECK(hipMemcpy(device->spheres, spheres_h, sizeof(t_sphere) * host->num_sp, hipMemcpyHostToDevice));
	if (host->num_pl > 0 && planes_h)
		CUDA_CHECK(hipMemcpy(device->planes, planes_h, sizeof(t_plane) * host->num_pl, hipMemcpyHostToDevice));
	if (host->num_cy > 0 && cylinders_h)
		CUDA_CHECK(hipMemcpy(device->cylinders, cylinders_h, sizeof(t_cylinder) * host->num_cy, hipMemcpyHostToDevice));
	if (host->num_co > 0 && cones_h)
		CUDA_CHECK(hipMemcpy(device->cones, cones_h, sizeof(t_cone) * host->num_co, hipMemcpyHostToDevice));
	if (host->num_lights > 0 && lights_h)
		CUDA_CHECK(hipMemcpy(device->lights, lights_h, sizeof(t_light_gpu) * host->num_lights, hipMemcpyHostToDevice));

	// 释放临时主机内存
	if (spheres_h) free(spheres_h);
	if (planes_h) free(planes_h);
	if (cylinders_h) free(cylinders_h);
	if (cones_h) free(cones_h);
	if (lights_h) free(lights_h);
}

extern "C" void render_with_cuda(t_minirt *rt)
{
    printf("Starting CUDA render...\n");
    
    // 1. 准备GPU场景数据
    t_minirt_gpu host_scene;
    t_minirt_gpu device_scene; // 这个结构体本身在CPU，但其指针成员指向GPU内存
    
    printf("Preparing GPU data...\n");
    prepare_gpu(rt, &host_scene, &device_scene);
    
    printf("Objects count: sp=%d, pl=%d, cy=%d, co=%d, lights=%d\n", 
           host_scene.num_sp, host_scene.num_pl, host_scene.num_cy, 
           host_scene.num_co, host_scene.num_lights);
    
    // 复制数量信息到device_scene
    device_scene.num_sp = host_scene.num_sp;
    device_scene.num_pl = host_scene.num_pl;
    device_scene.num_cy = host_scene.num_cy;
    device_scene.num_co = host_scene.num_co;
    device_scene.num_lights = host_scene.num_lights;
    device_scene.ambient = host_scene.ambient;
    device_scene.camera = host_scene.camera;
    
    printf("Copying scene to GPU...\n");
    // 将包含数量信息的场景结构体复制到GPU
    t_minirt_gpu* d_scene;
    CUDA_CHECK(hipMalloc(&d_scene, sizeof(t_minirt_gpu)));
    CUDA_CHECK(hipMemcpy(d_scene, &device_scene, sizeof(t_minirt_gpu), hipMemcpyHostToDevice));

    // 2. 在GPU上为输出图像分配内存
    int *d_pixel_buffer;
    CUDA_CHECK(hipMalloc(&d_pixel_buffer, WIDTH * HEIGHT * sizeof(int)));

    // 3. 设置内核启动参数
    dim3 threads_per_block(16, 16);
    dim3 num_blocks((WIDTH + 15) / 16, (HEIGHT + 15) / 16);

    // 4. 启动CUDA内核
    printf("Launching CUDA kernel...\n");
    render_kernel<<<num_blocks, threads_per_block>>>(d_pixel_buffer, d_scene);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize()); // 等待内核执行完成
    printf("Kernel finished.\n");

    // 5. 将渲染结果从GPU复制回CPU
    int *h_pixel_buffer = (int *)malloc(WIDTH * HEIGHT * sizeof(int));
    CUDA_CHECK(hipMemcpy(h_pixel_buffer, d_pixel_buffer, WIDTH * HEIGHT * sizeof(int), hipMemcpyDeviceToHost));

    // 6. 将结果复制到MLX的图像缓冲区
    memcpy(rt->img.addr, h_pixel_buffer, WIDTH * HEIGHT * sizeof(int));

    // 7. 释放所有内存
    free(h_pixel_buffer);
    CUDA_CHECK(hipFree(d_pixel_buffer));
    if (device_scene.spheres) CUDA_CHECK(hipFree(device_scene.spheres));
    if (device_scene.planes) CUDA_CHECK(hipFree(device_scene.planes));
    if (device_scene.cylinders) CUDA_CHECK(hipFree(device_scene.cylinders));
    if (device_scene.cones) CUDA_CHECK(hipFree(device_scene.cones));
    if (device_scene.lights) CUDA_CHECK(hipFree(device_scene.lights));
    CUDA_CHECK(hipFree(d_scene));
    
    printf("CUDA render completed.\n");
}

